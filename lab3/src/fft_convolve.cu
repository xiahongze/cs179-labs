#include "hip/hip_runtime.h"
/* CUDA blur
 * Kevin Yuh, 2014 */

#include <cstdio>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "fft_convolve.cuh"


/* 
Atomic-max function. You may find it useful for normalization.

We haven't really talked about this yet, but __device__ functions not
only are run on the GPU, but are called from within a kernel.

Source: 
http://stackoverflow.com/questions/17399119/
cant-we-use-atomic-operations-for-floating-point-variables-in-cuda
*/
__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}



__global__
void
cudaProdScaleKernel(const hipfftComplex *raw_data, const hipfftComplex *impulse_v, 
    hipfftComplex *out_data,
    int padded_length) {


    /* TODO: Implement the point-wise multiplication and scaling for the
    FFT'd input and impulse response. 

    Recall that these are complex numbers, so you'll need to use the
    appropriate rule for multiplying them. 

    Also remember to scale by the padded length of the signal
    (see the notes for Question 1).

    As in Assignment 1 and Week 1, remember to make your implementation
    resilient to varying numbers of threads.

    */

   uint thread_index = blockIdx.x * blockDim.x + threadIdx.x;
   float a, b, c, d;
   for (int i = thread_index; i < padded_length; i += blockDim.x * gridDim.x) {
        a = raw_data[i].x;
        b = raw_data[i].y;
        c = impulse_v[i].x;
        d = impulse_v[i].y;
        out_data[i].x = (a * c - b * d) / ((float) padded_length);
        out_data[i].y = (a * d + b * c) / ((float) padded_length);
   }
}

__global__
void
cudaMaximumKernel(hipfftComplex *out_data, float *max_abs_val,
    int padded_length) {

    /* TODO 2: Implement the maximum-finding.

    There are many ways to do this reduction, and some methods
    have much better performance than others. 

    For this section: Please explain your approach to the reduction,
    including why you chose the optimizations you did
    (especially as they relate to GPU hardware).

    You'll likely find the above atomicMax function helpful.
    (CUDA's atomicMax function doesn't work for floating-point values.)
    It's based on two principles:
        1) From Week 2, any atomic function can be implemented using
        atomic compare-and-swap.
        2) One can "represent" floating-point values as integers in
        a way that preserves comparison, if the sign of the two
        values is the same. (see http://stackoverflow.com/questions/
        29596797/can-the-return-value-of-float-as-int-be-used-to-
        compare-float-in-cuda)

    */

    // with the `extern` keyword, we can declare a shared memory array whose size is known at runtime
    // the size of the array is specified in the kernel call 
    extern __shared__ float sdata[];
    uint thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    float local_max = -INFINITY;
    for (int i = thread_index; i < padded_length; i += blockDim.x * gridDim.x) {
        local_max = max(local_max, out_data[i].x);
    }

    sdata[threadIdx.x] = local_max;
    __syncthreads();

    // reduction logic: each thread will compare its value with the value of the thread at the other end of the block
    // the thread with the smaller value will be discarded. This process is repeated until only one value remains.
    // the final value is stored in the first element of the shared memory array.
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            sdata[threadIdx.x] = max(sdata[threadIdx.x], sdata[threadIdx.x + i]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        // the first thread of each block will compare the value in the shared memory array with the value in the global memory array
        atomicMax(max_abs_val, sdata[0]);
    }
}

__global__
void
cudaDivideKernel(hipfftComplex *out_data, float *max_abs_val,
    int padded_length) {

    /* TODO 2: Implement the division kernel. Divide all
    data by the value pointed to by max_abs_val. 

    This kernel should be quite short.
    */

    uint thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = thread_index; i < padded_length; i += blockDim.x * gridDim.x) {
        out_data[i].x /= *max_abs_val;
        out_data[i].y /= *max_abs_val;
    }

}


void cudaCallProdScaleKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        const hipfftComplex *raw_data,
        const hipfftComplex *impulse_v,
        hipfftComplex *out_data,
        const unsigned int padded_length) {
        
    /* TODO: Call the element-wise product and scaling kernel. */

    cudaProdScaleKernel<<<blocks, threadsPerBlock>>>(raw_data, impulse_v, out_data, padded_length);

}

void cudaCallMaximumKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        hipfftComplex *out_data,
        float *max_abs_val,
        const unsigned int padded_length) {
        

    /* TODO 2: Call the max-finding kernel. */

    // following the cuda syntax, kernelName<<<numBlocks, numThreads, sharedMemSize>>>
    cudaMaximumKernel<<<blocks, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(out_data, max_abs_val, padded_length);
}


void cudaCallDivideKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        hipfftComplex *out_data,
        float *max_abs_val,
        const unsigned int padded_length) {
        
    /* TODO 2: Call the division kernel. */

    cudaDivideKernel<<<blocks, threadsPerBlock>>>(out_data, max_abs_val, padded_length);
}
