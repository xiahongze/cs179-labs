#include "hip/hip_runtime.h"
/**
 * CUDA-implemented utility functions & kernels needed by the neural net
 * @author Aadyot Bhatngar
 * @date April 22, 2018
 */

#include "utils.cuh"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

// CUDA block width
#define BW 1024

/**
 * Sets all entries in a device buffer of floats equal to a specified value.
 */
template<typename T> void cudaMemsetType(T *dev_ptr, T val, int n_vals)
{
    // thrust::device_ptr<T> thrust_dev_ptr(dev_ptr);
    // thrust::fill(thrust_dev_ptr, thrust_dev_ptr + n_vals, val);
    CUDA_CALL(hipMemset(dev_ptr, val, n_vals * sizeof(T)));
}

template<typename T> void printCudaArray(T *dev_ptr, int n_vals, const char *msg)
{
    std::cout << msg << std::endl;
    T *host_ptr = new T[n_vals];
    CUDA_CALL(hipMemcpy(host_ptr, dev_ptr, n_vals * sizeof(T),
        hipMemcpyDeviceToHost));

    for (int i = 0; i < n_vals; ++i)
        std::cout << host_ptr[i] << " ";
    std::cout << std::endl;

    delete[] host_ptr;
}


/**
 * Invokes a CUDA kernel to compute the average cross entropy between softmaxed
 * predictions pred_Y and ground truth true_Y.
 *
 * @param pred_Y predictions made by model (probability vectors)
 * @param true_Y true output values (one-hot vectors)
 * @param n number of predictions
 * @param c number of channels per prediction
 * @param h height of each prediction
 * @param w width of each prediction
 *
 * @return cross-entropy loss between pred_Y and true_Y
 */
float CrossEntropyLoss(float* pred_Y, float* true_Y, int n, int c, int h, int w)
{
    // Inialize loss on the device to be zero
    float loss, *d_loss;
    CUDA_CALL( hipMalloc(&d_loss, sizeof(float)) );
    cudaMemsetType<float>(d_loss, 0.0, 1);

    // Accumulate the total loss on the device by invoking a kernel
    int n_blocks = std::min(65535, (n * c * h * w + BW  - 1) / BW);
    // TODO (set 5): call CrossEntropyKernel
    CrossEntropyKernel<<<n_blocks, BW, BW * sizeof(float)>>>(pred_Y, true_Y,
        d_loss, n, c, h, w);

    // Copy back the accumulated loss on the device back to the host
    CUDA_CALL( hipMemcpy(&loss, d_loss, sizeof(float), hipMemcpyDeviceToHost) );

    CUDA_CALL( hipFree(d_loss) );
    // Return the average loss
    return loss;
    // return computeCrossEntropyLoss(pred_Y, true_Y, n, c, h, w);
}

/**
* Invokes a CUDA kernel to compute the average accuracy of softmaxed predictions
* pred_Y, given ground truth true_Y.
*
* @param pred_Y predictions made by model (probability vectors)
* @param true_Y true output values (one-hot vectors)
* @param n number of predictions
* @param c number of channels per prediction
* @param h height of each prediction
* @param w width of each prediction
*
* @return proportion of n for which the maximum entry in pred_Y (most probable
*         class predicted) is the same as the one entry in true_Y (true class)
*/
float SoftThresholdAccuracy(float* pred_Y, float* true_Y,
    int n, int c, int h, int w)
{
    // Initialize the accuracy on the device to be zero
    float acc, *d_acc;
    CUDA_CALL( hipMalloc(&d_acc, sizeof(float)) );
    cudaMemsetType<float>(d_acc, 0.0, 1);

    // Accumulate the total loss on the device by invoking a kernel
    int n_blocks = std::min(65535, (n * c * h * w + BW - 1) / BW);
    SoftThresholdAccKernel<<<n_blocks, BW, BW * sizeof(float)>>>(pred_Y, true_Y,
        d_acc, n, c, h, w);

    // Copy back the accumulated accuracy on the device back to the host
    CUDA_CALL(hipMemcpy(&acc, d_acc, sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CALL(hipFree(d_acc));

    // Return the average accuracy
    return acc / static_cast<float>(n);
}

float computeCrossEntropyLoss(float* dev_pred_Y, float* dev_true_Y, int n, int c, int h, int w)
{
    // Copy the predictions and ground truth to the host
    float *pred_Y = new float[n * c * h * w];
    float *true_Y = new float[n * c * h * w];
    CUDA_CALL(hipMemcpy(pred_Y, dev_pred_Y, n * c * h * w * sizeof(float),
        hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(true_Y, dev_true_Y, n * c * h * w * sizeof(float),
        hipMemcpyDeviceToHost));
    float loss = 0.0;
    for (int i = 0; i < n * c * h * w; ++i)
    {
        loss -= log(pred_Y[i]) * true_Y[i];
    }
    return loss / static_cast<float>(n);
}



/**
 * Kernel to compute cross-entropy between pred_Y and true_Y as described by
 * {\link CrossEntropyLoss}.
 */
__global__ void CrossEntropyKernel(float* pred_Y, float* true_Y, float *loss,
    int n, int c, int h, int w)
{
    extern __shared__ float shmem[];

    // TODO (set 5): use a parallel reduction to compute cross-entropy between
    //               pred_Y and true_Y, i.e. -sum( log(pred_Y[i]) * true_Y[i] ),
    //               where i ranges from 0 to (n*c*h*w) - 1

    // have each thread in each block accumulate some of the total loss in
    // shared memory
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    shmem[threadIdx.x] = 0.0;
    for (; idx < n * c * h * w; idx += blockDim.x * gridDim.x)
    {
        shmem[threadIdx.x] -= log(pred_Y[idx]) * true_Y[idx];
    }

    __syncthreads();

    // do a reduction to sum up all of the loss components in this block's
    // shared memory
    for (unsigned s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
            shmem[threadIdx.x] += shmem[threadIdx.x + s];
        __syncthreads();
    }

    // atomically add the accumulated loss per block into the global accumulator
    if (threadIdx.x == 0)
        atomicAdd(loss, shmem[0] / static_cast<float>(n));
}

/**
 * Kernel to compute accuracy of pred_Y given ground truth true_Y as described
 * by {\link SoftThresholdAccuracy}.
 */
__global__ void SoftThresholdAccKernel(float* pred_Y, float* true_Y, float* acc,
    int n, int c, int h, int w)
{
    extern __shared__ float shmem[];
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned tid = threadIdx.x;

    // have each thread in each block accumulate some of the total loss in
    // shared memory
    shmem[tid] = 0.0;
    for (; idx < n; idx += blockDim.x * gridDim.x)
    {
        unsigned idx_cur = idx * c * h * w;

        // Determine which copmonent/element of the current prediction vector
        // and its corresponding ground truth is largest
        unsigned argmax_pred = 0, argmax_true = 0;
        for (unsigned j = 0; j < c * h * w; ++j)
        {
            if (pred_Y[idx_cur + argmax_pred] < pred_Y[idx_cur + j])
                argmax_pred = j;

            if (true_Y[idx_cur + argmax_true] < true_Y[idx_cur + j])
                argmax_true = j;
        }

        // If we were correct, add 1 to the accuracy count
        if (argmax_pred == argmax_true)
            shmem[tid] += 1.0;
    }
    __syncthreads();

    // do a reduction to sum up all of the accuracy components in this block's
    // shared memory
    for (unsigned s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
            shmem[tid] += shmem[tid + s];
        __syncthreads();
    }

    // atomically add the accumulated accuracy per block into the global accumulator
    if (tid == 0) atomicAdd(acc, shmem[tid]);
}
